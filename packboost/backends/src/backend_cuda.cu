#include "hip/hip_runtime.h"
#include "backend.hpp"

#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>

#include <cstdint>
#include <cstring>
#include <stdexcept>
#include <string>
#include <vector>

namespace py = pybind11;

namespace packboost {

namespace {

constexpr int WARP_SIZE = 32;

inline void check(hipError_t status, const char* msg) {
    if (status != hipSuccess) {
        throw std::runtime_error(std::string(msg) + ": " + hipGetErrorString(status));
    }
}

__inline__ __device__ int lane_id() {
    int id;
    asm volatile("mov.s32 %0, %laneid;" : "=r"(id));
    return id;
}

__global__ void histogram_kernel(
    const uint8_t* __restrict__ bins,
    const float* __restrict__ gradients,
    const float* __restrict__ hessians,
    const int16_t* __restrict__ era_inverse,
    int n_rows,
    int n_features,
    int max_bins,
    int n_eras,
    float* __restrict__ out_grad,
    float* __restrict__ out_hess,
    int32_t* __restrict__ out_count) {

    extern __shared__ unsigned char shared_raw[];
    float* shared_grad = reinterpret_cast<float*>(shared_raw);
    float* shared_hess = shared_grad + max_bins * n_eras;
    int32_t* shared_count = reinterpret_cast<int32_t*>(shared_hess + max_bins * n_eras);

    const int feature = blockIdx.x;
    // zero shared memory
    for (int idx = threadIdx.x; idx < max_bins * n_eras; idx += blockDim.x) {
        shared_grad[idx] = 0.0f;
        shared_hess[idx] = 0.0f;
        shared_count[idx] = 0;
    }
    __syncthreads();

    for (int row = threadIdx.x; row < n_rows; row += blockDim.x) {
        const int era = static_cast<int>(era_inverse[row]);
        if (era < 0 || era >= n_eras) continue;
        const uint8_t bin = bins[row * n_features + feature];
        if (bin >= max_bins) continue;
        const int idx = bin * n_eras + era;
        atomicAdd(shared_grad + idx, gradients[row]);
        atomicAdd(shared_hess + idx, hessians[row]);
        atomicAdd(shared_count + idx, 1);
    }
    __syncthreads();

    float* grad_out = out_grad + feature * max_bins * n_eras;
    float* hess_out = out_hess + feature * max_bins * n_eras;
    int32_t* count_out = out_count + feature * max_bins * n_eras;

    for (int idx = threadIdx.x; idx < max_bins * n_eras; idx += blockDim.x) {
        atomicAdd(grad_out + idx, shared_grad[idx]);
        atomicAdd(hess_out + idx, shared_hess[idx]);
        atomicAdd(count_out + idx, shared_count[idx]);
    }
}

void launch_histogram_kernel(
    const uint8_t* bins,
    const float* gradients,
    const float* hessians,
    const int16_t* era_inverse,
    int n_rows,
    int n_features,
    int max_bins,
    int n_eras,
    float* out_grad,
    float* out_hess,
    int32_t* out_count) {

    const dim3 grid(n_features);
    const int threads = 256;
    const std::size_t shared = static_cast<std::size_t>(max_bins) * n_eras * (2 * sizeof(float) + sizeof(int32_t));
    histogram_kernel<<<grid, threads, shared>>>(
        bins, gradients, hessians, era_inverse,
        n_rows, n_features, max_bins, n_eras,
        out_grad, out_hess, out_count);
    check(hipDeviceSynchronize(), "histogram_kernel launch failed");
}

}  // namespace

HistogramBuffers build_histograms_cuda(
    const uint8_t* bins,
    const float* gradients,
    const float* hessians,
    const int16_t* era_inverse,
    std::size_t n_rows,
    std::size_t n_features,
    int max_bins,
    int n_eras) {

    const std::size_t row_bytes = n_rows * n_features * sizeof(uint8_t);
    const std::size_t vec_bytes = n_rows * sizeof(float);
    const std::size_t era_bytes = n_rows * sizeof(int16_t);
    const std::size_t hist_size = n_features * max_bins * n_eras;
    const std::size_t hist_bytes_f = hist_size * sizeof(float);
    const std::size_t hist_bytes_i = hist_size * sizeof(int32_t);

    uint8_t* d_bins = nullptr;
    float* d_grad = nullptr;
    float* d_hess = nullptr;
    int16_t* d_era = nullptr;
    float* d_out_grad = nullptr;
    float* d_out_hess = nullptr;
    int32_t* d_out_count = nullptr;

    check(hipMalloc(&d_bins, row_bytes), "hipMalloc bins");
    check(hipMalloc(&d_grad, vec_bytes), "hipMalloc grad");
    check(hipMalloc(&d_hess, vec_bytes), "hipMalloc hess");
    check(hipMalloc(&d_era, era_bytes), "hipMalloc era");
    check(hipMalloc(&d_out_grad, hist_bytes_f), "hipMalloc out_grad");
    check(hipMalloc(&d_out_hess, hist_bytes_f), "hipMalloc out_hess");
    check(hipMalloc(&d_out_count, hist_bytes_i), "hipMalloc out_count");
    check(hipMemset(d_out_grad, 0, hist_bytes_f), "hipMemset out_grad");
    check(hipMemset(d_out_hess, 0, hist_bytes_f), "hipMemset out_hess");
    check(hipMemset(d_out_count, 0, hist_bytes_i), "hipMemset out_count");

    check(hipMemcpy(d_bins, bins, row_bytes, hipMemcpyHostToDevice), "hipMemcpy bins");
    check(hipMemcpy(d_grad, gradients, vec_bytes, hipMemcpyHostToDevice), "hipMemcpy grad");
    check(hipMemcpy(d_hess, hessians, vec_bytes, hipMemcpyHostToDevice), "hipMemcpy hess");
    check(hipMemcpy(d_era, era_inverse, era_bytes, hipMemcpyHostToDevice), "hipMemcpy era");

    launch_histogram_kernel(
        d_bins,
        d_grad,
        d_hess,
        d_era,
        static_cast<int>(n_rows),
        static_cast<int>(n_features),
        max_bins,
        n_eras,
        d_out_grad,
        d_out_hess,
        d_out_count);

    HistogramBuffers buffers;
    buffers.grad.resize(hist_size);
    buffers.hess.resize(hist_size);
    buffers.count.resize(hist_size);

    check(hipMemcpy(buffers.grad.data(), d_out_grad, hist_bytes_f, hipMemcpyDeviceToHost), "hipMemcpy back grad");
    check(hipMemcpy(buffers.hess.data(), d_out_hess, hist_bytes_f, hipMemcpyDeviceToHost), "hipMemcpy back hess");
    check(hipMemcpy(buffers.count.data(), d_out_count, hist_bytes_i, hipMemcpyDeviceToHost), "hipMemcpy back count");

    hipFree(d_bins);
    hipFree(d_grad);
    hipFree(d_hess);
    hipFree(d_era);
    hipFree(d_out_grad);
    hipFree(d_out_hess);
    hipFree(d_out_count);

    return buffers;
}

FrontierEvalResult evaluate_frontier_cuda(
    const uint8_t* bins,
    const int32_t* node_indices,
    const int32_t* node_offsets,
    const int32_t* node_era_offsets,
    const int32_t* era_group_eras,
    const int32_t* era_group_offsets,
    const int32_t* feature_subset,
    const float* gradients,
    const float* hessians,
    std::size_t n_rows,
    std::size_t n_features_total,
    std::size_t n_nodes,
    std::size_t n_features_subset,
    int max_bins,
    double lambda_l2,
    double lambda_dro,
    int min_samples_leaf,
    double direction_weight,
    int era_tile_size) {

    // TODO: implement full CUDA frontier evaluation once kernels are available.
    // For now, fall back to the tuned CPU implementation so the interface stays
    // functional when only the CUDA extension is built.
    return evaluate_frontier_cpu(
        bins,
        node_indices,
        node_offsets,
        node_era_offsets,
        era_group_eras,
        era_group_offsets,
        feature_subset,
        gradients,
        hessians,
        n_rows,
        n_features_total,
        n_nodes,
        n_features_subset,
        max_bins,
        lambda_l2,
        lambda_dro,
        min_samples_leaf,
        direction_weight,
        era_tile_size);
}

}  // namespace packboost

py::tuple cuda_histogram_binding(
    py::array_t<uint8_t, py::array::c_style | py::array::forcecast> bins,
    py::array_t<float, py::array::c_style | py::array::forcecast> gradients,
    py::array_t<float, py::array::c_style | py::array::forcecast> hessians,
    py::array_t<int16_t, py::array::c_style | py::array::forcecast> era_inverse,
    int max_bins,
    int n_eras) {

    py::buffer_info bins_info = bins.request();
    py::buffer_info grad_info = gradients.request();
    py::buffer_info hess_info = hessians.request();
    py::buffer_info era_info = era_inverse.request();

    if (bins_info.ndim != 2) {
        throw std::invalid_argument("bins must be 2D");
    }

    const std::size_t n_rows = static_cast<std::size_t>(bins_info.shape[0]);
    const std::size_t n_features = static_cast<std::size_t>(bins_info.shape[1]);

    auto buffers = packboost::build_histograms_cuda(
        static_cast<uint8_t*>(bins_info.ptr),
        static_cast<float*>(grad_info.ptr),
        static_cast<float*>(hess_info.ptr),
        static_cast<int16_t*>(era_info.ptr),
        n_rows,
        n_features,
        max_bins,
        n_eras);

    std::vector<py::ssize_t> shape = {
        static_cast<py::ssize_t>(n_features),
        static_cast<py::ssize_t>(max_bins),
        static_cast<py::ssize_t>(n_eras),
    };
    py::array_t<float> grad_arr(shape);
    py::array_t<float> hess_arr(shape);
    py::array_t<int32_t> count_arr(shape);
    std::memcpy(grad_arr.mutable_data(), buffers.grad.data(), buffers.grad.size() * sizeof(float));
    std::memcpy(hess_arr.mutable_data(), buffers.hess.data(), buffers.hess.size() * sizeof(float));
    std::memcpy(count_arr.mutable_data(), buffers.count.data(), buffers.count.size() * sizeof(int32_t));

    return py::make_tuple(std::move(grad_arr), std::move(hess_arr), std::move(count_arr));
}

py::tuple cuda_frontier_evaluate_binding(
    py::array_t<uint8_t, py::array::c_style | py::array::forcecast> bins,
    py::array_t<int32_t, py::array::c_style | py::array::forcecast> node_indices,
    py::array_t<int32_t, py::array::c_style | py::array::forcecast> node_offsets,
    py::array_t<int32_t, py::array::c_style | py::array::forcecast> node_era_offsets,
    py::array_t<int32_t, py::array::c_style | py::array::forcecast> era_group_eras,
    py::array_t<int32_t, py::array::c_style | py::array::forcecast> era_group_offsets,
    py::array_t<int32_t, py::array::c_style | py::array::forcecast> feature_subset,
    py::array_t<float, py::array::c_style | py::array::forcecast> gradients,
    py::array_t<float, py::array::c_style | py::array::forcecast> hessians,
    int max_bins,
    int n_eras_total,
    double lambda_l2,
    double lambda_dro,
    int min_samples_leaf,
    double direction_weight,
    int era_tile_size) {

    py::buffer_info bins_info = bins.request();
    py::buffer_info idx_info = node_indices.request();
    py::buffer_info offsets_info = node_offsets.request();
    py::buffer_info node_era_info = node_era_offsets.request();
    py::buffer_info era_group_info = era_group_eras.request();
    py::buffer_info era_group_offsets_info = era_group_offsets.request();
    py::buffer_info feat_info = feature_subset.request();
    py::buffer_info grad_info = gradients.request();
    py::buffer_info hess_info = hessians.request();

    if (bins_info.ndim != 2) {
        throw std::invalid_argument("bins must be 2D");
    }

    const std::size_t n_rows = static_cast<std::size_t>(bins_info.shape[0]);
    const std::size_t n_features_total = static_cast<std::size_t>(bins_info.shape[1]);
    const std::size_t n_nodes = static_cast<std::size_t>(offsets_info.shape[0] - 1);
    const std::size_t n_features_subset = static_cast<std::size_t>(feat_info.shape[0]);

    auto result = packboost::evaluate_frontier_cuda(
        static_cast<uint8_t*>(bins_info.ptr),
        static_cast<int32_t*>(idx_info.ptr),
        static_cast<int32_t*>(offsets_info.ptr),
        static_cast<int32_t*>(node_era_info.ptr),
        static_cast<int32_t*>(era_group_info.ptr),
        static_cast<int32_t*>(era_group_offsets_info.ptr),
        static_cast<int32_t*>(feat_info.ptr),
        static_cast<float*>(grad_info.ptr),
        static_cast<float*>(hess_info.ptr),
        n_rows,
        n_features_total,
        n_nodes,
        n_features_subset,
        max_bins,
        n_eras_total,
        lambda_l2,
        lambda_dro,
        min_samples_leaf,
        direction_weight,
        era_tile_size);

    std::vector<py::ssize_t> vec_shape = {static_cast<py::ssize_t>(n_nodes)};
    py::array_t<int32_t> feature_arr(vec_shape);
    py::array_t<int32_t> threshold_arr(vec_shape);
    py::array_t<float> score_arr(vec_shape);
    py::array_t<float> agreement_arr(vec_shape);
    py::array_t<float> left_value_arr(vec_shape);
    py::array_t<float> right_value_arr(vec_shape);
    py::array_t<float> base_value_arr(vec_shape);

    std::vector<py::ssize_t> offset_shape = {static_cast<py::ssize_t>(n_nodes + 1)};
    py::array_t<int32_t> left_offsets_arr(offset_shape);
    py::array_t<int32_t> right_offsets_arr(offset_shape);

    std::vector<py::ssize_t> left_idx_shape = {static_cast<py::ssize_t>(result.left_indices.size())};
    std::vector<py::ssize_t> right_idx_shape = {static_cast<py::ssize_t>(result.right_indices.size())};
    py::array_t<int32_t> left_indices_arr(left_idx_shape);
    py::array_t<int32_t> right_indices_arr(right_idx_shape);

    std::memcpy(feature_arr.mutable_data(), result.best_feature.data(), n_nodes * sizeof(int32_t));
    std::memcpy(threshold_arr.mutable_data(), result.best_threshold.data(), n_nodes * sizeof(int32_t));
    std::memcpy(score_arr.mutable_data(), result.score.data(), n_nodes * sizeof(float));
    std::memcpy(agreement_arr.mutable_data(), result.agreement.data(), n_nodes * sizeof(float));
    std::memcpy(left_value_arr.mutable_data(), result.left_value.data(), n_nodes * sizeof(float));
    std::memcpy(right_value_arr.mutable_data(), result.right_value.data(), n_nodes * sizeof(float));
    std::memcpy(base_value_arr.mutable_data(), result.base_value.data(), n_nodes * sizeof(float));
    std::memcpy(left_offsets_arr.mutable_data(), result.left_offsets.data(), (n_nodes + 1) * sizeof(int32_t));
    std::memcpy(right_offsets_arr.mutable_data(), result.right_offsets.data(), (n_nodes + 1) * sizeof(int32_t));
    if (!result.left_indices.empty()) {
        std::memcpy(left_indices_arr.mutable_data(), result.left_indices.data(), result.left_indices.size() * sizeof(int32_t));
    }
    if (!result.right_indices.empty()) {
        std::memcpy(right_indices_arr.mutable_data(), result.right_indices.data(), result.right_indices.size() * sizeof(int32_t));
    }

    return py::make_tuple(
        feature_arr,
        threshold_arr,
        score_arr,
        agreement_arr,
        left_value_arr,
        right_value_arr,
        base_value_arr,
        left_offsets_arr,
        right_offsets_arr,
        left_indices_arr,
        right_indices_arr);
}
